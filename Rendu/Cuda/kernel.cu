#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>


//Cuda kernel function to add vector , based on Course CM 
//Global point d'entrée dans GPU a partir de CPU 
__global__ void vec_add(const float *a, const float *b, float *c, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; // i = indice globale du thread , blockIdx: indice de bloc dans la grille , blockDim : nombre de thread dans le bloc et threadIdx 
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

// Version CPU pour verifier 
void vec_add_cpu(const float *a, const float *b, float *c, int N){
    for(int i=0; i<N ; i++){
        c[i] = a[i] + b[i];
    }
}

// Vérifie que les vecteurs sont identiques
bool verify(const float* ref, const float* res, int N, int epsilon) {
    for (int i = 0; i < N; i++) {
        if (fabs(ref[i] - res[i]) > epsilon) return false;
    }
    return true;
}


int main() {
    int N = 1<<25;  
    size_t size = N * sizeof(float);
    
    // Allocation mémoire sur l'hôte
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *h_C_ref = (float*)malloc(size);

    // Initialisation des vecteurs avec des valeurs aléatoires
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // CPU reference
    vec_add_cpu(h_A, h_B, h_C_ref, N);

    //Allocation GPU 
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);


     // Chrono CUDA
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);


    // Copie des données depuis l'hôte vers le device ( CPU vers GPU )
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_h2d;
    hipEventElapsedTime(&time_h2d, start, stop);



    // Lancer le kernel avec un nombre de threads par bloc et de blocs
    // Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start);
    vec_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_kernel;
    hipEventElapsedTime(&time_kernel, start, stop);


      // Transfert vers CPU
      hipEventRecord(start);
      hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float time_d2h;
      hipEventElapsedTime(&time_d2h, start, stop);

          // Verify results
    if (verify(h_C_ref, h_C, N, 1e-5f)) {
        printf("CPU addition verified successfully!\n");
    } else {
        printf("Error in CPU addition!\n");
    }

    // Output the times
    printf("CPU Time: %f seconds\n", time_h2d / 1000);
    printf("GPU Kernel Time: %f seconds\n", time_kernel / 1000);
    printf("Data Transfer Time (D->H): %f seconds\n", time_d2h / 1000);
  
      // Nettoyage
      hipFree(d_A); hipFree(d_B); hipFree(d_C);
      free(h_A); free(h_B); free(h_C); free(h_C_ref);
  

    return 0;
}