#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>
#include <cstdlib>

// CUDA kernel function to add vectors
__global__ void vec_add(const float *a, const float *b, float *c, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread index
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

// CPU version of vector addition for reference
void vec_add_cpu(const float *a, const float *b, float *c, int N) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

// Verifies if two vectors are identical within a given tolerance
bool verify(const float* ref, const float* res, int N, float epsilon = 1e-5f) {
    for (int i = 0; i < N; i++) {
        if (fabs(ref[i] - res[i]) > epsilon) {
            return false; // Not equal
        }
    }
    return true; // Equal
}

int main() {
    // Open file for writing results
    FILE *file = fopen("timing_results.csv", "w");
    if (file == NULL) {
        printf("Error opening file for writing results.\n");
        return -1;
    }
    fprintf(file, "N,CPU_Time,GPU_Time\n");

    // Loop over different values of N
    for (int i = 0; i < 15; i++) {
        int N = 1 << (i + 10);  // Varying N from 2^10 to 2^25
        size_t size = N * sizeof(float);

        // Allocate memory on the host
        float *h_A = (float*)malloc(size);
        float *h_B = (float*)malloc(size);
        float *h_C = (float*)malloc(size);
        float *h_C_ref = (float*)malloc(size);

        // Initialize vectors with random values
        for (int i = 0; i < N; i++) {
            h_A[i] = (float)rand() / RAND_MAX;
            h_B[i] = (float)rand() / RAND_MAX;
        }

        // CPU reference
        clock_t start_cpu, end_cpu;
        double cpu_time_used;
        start_cpu = clock();
        vec_add_cpu(h_A, h_B, h_C_ref, N);
        end_cpu = clock();  // Fin du chronométrage
        cpu_time_used = ((double) (end_cpu - start_cpu)) / CLOCKS_PER_SEC;

        // Allocate GPU memory
        float *d_A, *d_B, *d_C;
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        hipMalloc(&d_C, size);

        // Chrono CUDA
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Copy data from host to device (CPU -> GPU)
        hipEventRecord(start);
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time_h2d;
        hipEventElapsedTime(&time_h2d, start, stop);

        // Launch the kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        hipEventRecord(start);
        vec_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time_kernel;
        hipEventElapsedTime(&time_kernel, start, stop);

        // Copy back data to host (GPU -> CPU)
        hipEventRecord(start);
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time_d2h;
        hipEventElapsedTime(&time_d2h, start, stop);

        // Verify results
        if (verify(h_C_ref, h_C, N, 1e-5f)) {
            printf("CPU addition verified successfully for N = %d!\n", N);
        } else {
            printf("Error in CPU addition for N = %d!\n", N);
        }

        // Output the times to the file
        fprintf(file, "%d,%f,%f\n", N,  cpu_time_used, time_kernel / 1000);

        // Free memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        free(h_C_ref);
    }

    // Close the file
    fclose(file);
    printf("Timing results have been written to 'timing_results.csv'.\n");

    return 0;
}
